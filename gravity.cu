#include "hip/hip_runtime.h"
#include <iostream>
#include <SDL2/SDL.h>
#include <cmath>
#include <cstdlib>
#include <ctime>

const int SCREEN_WIDTH = 900;
const int SCREEN_HEIGHT = 900;
const float dt = 0.31f;
const float G = 6.67408e-11;
const float CENTRAL_MASS = 26.44e12;

struct Star {
    float x, y;
    float mass;
    float vx, vy;
    float brightness, opacity; 
};

struct DarkMatter {
    float x, y;
    float mass;
    float vx, vy;
};

__device__ float calculateEscapeVelocity(Star* star, float distance) {
    return sqrtf(2 * G * star->mass / distance);
}

__global__ void updateStars(Star* stars, int numStars, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float MAX_FORCE = 1e-5f;
    const float MIN_BRIGHTNESS = 0.1f;
    if (i < numStars) {
        for (int j = 0; j < numStars; j++) {
            if (i != j) {
                float dx = stars[j].x - stars[i].x;
                float dy = stars[j].y - stars[i].y;
                float dist = sqrtf(dx * dx + dy * dy) + 1e-5f;
                float escapeVelocity = calculateEscapeVelocity(&stars[j], dist);
                float distanceSquared = dx * dx + dy * dy + 1e-5f;
                float force = G * stars[j].mass * stars[i].mass / (distanceSquared);
                stars[i].brightness += MAX_FORCE / ((SCREEN_WIDTH / 2) * (SCREEN_HEIGHT / 2) );
                stars[i].brightness = min(max(stars[i].brightness, MIN_BRIGHTNESS), 1.0f);
                float ax = force * (dx / sqrt(distanceSquared)) / stars[i].mass;
                float ay = force * (dy / sqrt(distanceSquared)) / stars[i].mass;
                stars[i].vx += ax * dt;
                stars[i].vy += ay * dt;
            }
        }
        stars[i].x += stars[i].vx * dt / 2;
        stars[i].y += stars[i].vy * dt / 2;
    }
}

int main() {
    SDL_Init(SDL_INIT_VIDEO);
    SDL_Window* window = SDL_CreateWindow("Galaxy Simulation", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    const float tiltAngle = M_PI / 8; // 30 degrees, for example
    const int numStars = 25500;
    Star* stars;
    hipMallocManaged(&stars, numStars * sizeof(Star));

    srand(static_cast<unsigned int>(time(NULL)));
    stars[0].x = SCREEN_WIDTH / 2;
    stars[0].y = SCREEN_HEIGHT / 2;
    stars[0].vx = 0;
    stars[0].vy = 0;
    stars[0].mass = CENTRAL_MASS;

    int offsetX = 0, offsetY = 0;
    for (int i = 1; i < numStars; i++) {
        float radius = static_cast<float>(rand()) / RAND_MAX * (SCREEN_WIDTH / 2) / 2;
        float angle = static_cast<float>(rand()) / RAND_MAX * 2.0f * M_PI;
        stars[i].opacity = static_cast<float>(rand()) / RAND_MAX;
        stars[i].x = SCREEN_WIDTH / 2 + radius * cosf(angle) - offsetX;
        stars[i].y = SCREEN_HEIGHT / 2 + radius * sinf(angle) - offsetY;

        float velocity = sqrt(G * CENTRAL_MASS / radius);
        stars[i].vx = -velocity * sinf(angle); 
        stars[i].vy = velocity * cosf(angle);
        stars[i].mass = 1 + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (1e7 - 1e6)));
    }

    const int numThreads = 512;
    const int numBlocks = (numStars + numThreads - 1) / numThreads;
    float zoom = 1.0f; 
    bool quit = false;
    bool isDragging = false;
    int lastMouseX, lastMouseY;
    while (!quit) {
        SDL_Event event;
        while (SDL_PollEvent(&event)) {
        switch (event.type) {
                        case SDL_QUIT:
                            quit = true;
                            break;
                        case SDL_MOUSEBUTTONDOWN:
                            if (event.button.button == SDL_BUTTON_LEFT) {
                                isDragging = true;
                                lastMouseX = event.button.x;
                                lastMouseY = event.button.y;
                            }
                            break;
                        case SDL_MOUSEBUTTONUP:
                            if (event.button.button == SDL_BUTTON_LEFT) {
                                isDragging = false;
                            }
                            break;
                        case SDL_MOUSEMOTION:
                            if (isDragging) {
                                int mouseX, mouseY;
                                SDL_GetMouseState(&mouseX, &mouseY);

                                offsetX += mouseX - lastMouseX;
                                offsetY += mouseY - lastMouseY;

                                lastMouseX = mouseX;
                                lastMouseY = mouseY;
                            }
                        case SDL_MOUSEWHEEL:
                            case SDL_KEYDOWN:
                                if (event.key.keysym.sym == SDLK_q) {
                                    zoom *= 1.01f; // zoom in
                                }
                                else if (event.key.keysym.sym == SDLK_a) {
                                    zoom /= 1.01f; // zoom out
                                }
                                break;
                        }
                    }

            updateStars<<<numBlocks, numThreads>>>(stars, numStars, dt);
            hipDeviceSynchronize();

            SDL_SetRenderDrawColor(renderer, 22, 22, 22, 255); 
            SDL_RenderClear(renderer);


        for (int i = 0; i < numStars; i++) 
        {        
            float tiltedY = stars[i].y + tan(tiltAngle) * stars[i].x;
            SDL_SetRenderDrawColor(renderer, 255 / stars[i].brightness, 255, 102, stars[i].opacity * 255); 
            SDL_RenderDrawPoint(renderer, (stars[i].x + (offsetX - 600) + tiltedY) * zoom, (stars[i].y + offsetY) * zoom);
        }
        SDL_RenderPresent(renderer);
    }

    hipFree(stars);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
 


